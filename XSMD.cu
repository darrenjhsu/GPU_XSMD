
#include <stdio.h>
#include <math.h>
#include "kernel.cu"
#include "XSMD.hh"
#include "mol_param.hh"
#include "env_param.hh"
#include "scat_param.hh"



void XSMD_calc (float *coord, float *Force) {

    // In this code pointers with d_ are device pointers. 

    // Declare local pointers //
    // The calculated scattering pattern for this snapshot.
    float *S_calc;

    // Declare cuda pointers //
    // The prefactor, coordinates, force to be returned, and force field table (num_q * num_ele)
    float *d_Aq, *d_coord, *d_Force, *d_FF;
    // List of what element each atom is.
    int *d_Ele;
    // q vector, reference scattering pattern and measured difference pattern to fit.
    float *d_q_S_ref_dS, *d_S_calc;
    // Some intermediate matrices
    float *d_S_calcc, *d_f_ptxc, *d_f_ptyc, *d_f_ptzc;
    // Displacement of the coordinates (N x N matrices)
    float *d_dx, *d_dy, *d_dz;
    float *d_V, *d_r2;
    float *d_WK;
    int *d_close_flag, *d_close_num, *d_close_idx;
    float *d_vdW;
    float *d_FF_table;
    float *d_surf, *d_surf_grad;
    float *d_c2;

    // set various memory chunk sizes
    int size_coord = 3 * num_atom * sizeof(float);
    int size_atom = num_atom * sizeof(int);
    int size_atom2 = num_atom2 * sizeof(int);
    int size_atom2f = num_atom2 * sizeof(float);
    int size_atomxatom = num_atom * num_atom * sizeof(float);
    int size_atom2xatom2 = num_atom2 * num_atom2 * sizeof(int);
    int size_q = num_q * sizeof(float); 
    int size_FF = num_ele * num_q * sizeof(float);
    int size_qxatom2 = num_q2 * num_atom2 * sizeof(float); // check if overflow
    int size_FF_table = (num_ele+1) * num_q * sizeof(float);
    int size_surf = num_atom * num_raster * 3 * sizeof(float);
    int size_WK = 11 * num_ele * sizeof(float);
    int size_vdW = (num_ele+1) * sizeof(float);
    int size_c2 = 9 * sizeof(float);

    // Allocate local memories
    //Force = (float *)malloc(size_coord);
    S_calc = (float *)malloc(size_q);

    /*for (int ii = 0; ii < num_atom; ii ++) {
        printf("%.3f %.3f %.3f \n", coord[3*ii], coord[3*ii+1], coord[3*ii+2]);
    }*/
    // Allocate cuda memories
    hipMalloc((void **)&d_Aq,     size_q);
    hipMalloc((void **)&d_coord,  size_coord); // 40 KB
    hipMalloc((void **)&d_Force,  size_coord); // 40 KB
    hipMalloc((void **)&d_FF,     size_FF);  // 10 KB ?
    hipMalloc((void **)&d_Ele,    size_atom);
    hipMalloc((void **)&d_q_S_ref_dS, 3 * size_q);
    hipMalloc((void **)&d_S_calc, size_q); // Will be computed on GPU
    hipMalloc((void **)&d_f_ptxc, size_qxatom2);
    hipMalloc((void **)&d_f_ptyc, size_qxatom2);
    hipMalloc((void **)&d_f_ptzc, size_qxatom2);
    hipMalloc((void **)&d_S_calcc, size_qxatom2);
    hipMalloc((void **)&d_V, size_atom2f);
    hipMalloc((void **)&d_dx, size_atomxatom);
    hipMalloc((void **)&d_dy, size_atomxatom);
    hipMalloc((void **)&d_dz, size_atomxatom);
    hipMalloc((void **)&d_r2, size_atomxatom);
    hipMalloc((void **)&d_close_flag, size_atom2xatom2);
    hipMalloc((void **)&d_close_num, size_atom2);
    hipMalloc((void **)&d_close_idx, size_atom2xatom2);
    hipMalloc((void **)&d_vdW, size_vdW);
    hipMalloc((void **)&d_FF_table, size_FF_table);
    hipMalloc((void **)&d_WK, size_WK);
    hipMalloc((void **)&d_surf, size_surf);
    hipMalloc((void **)&d_surf_grad, size_coord);
    hipMalloc((void **)&d_c2, size_c2);
    // Initialize some matrices
    hipMemset(d_close_flag, 0, size_qxatom2);
    hipMemset(d_Force, 0.0, size_coord);
    hipMemset(d_Aq, 0.0, size_q);
    hipMemset(d_S_calc, 0.0, size_q);
    hipMemset(d_f_ptxc,0.0, size_qxatom2);
    hipMemset(d_f_ptyc,0.0, size_qxatom2);   
    hipMemset(d_f_ptzc,0.0, size_qxatom2);
    hipMemset(d_S_calcc,0.0, size_qxatom2);
    hipMemset(d_close_num, 0, size_atom2);
    hipMemset(d_close_idx, 0, size_atom2xatom2);
    hipMemset(d_surf, 0.0, size_surf);
    hipMemset(d_surf_grad, 0.0, size_coord);
    // Copy necessary data
    hipMemcpy(d_coord, coord, size_coord,    hipMemcpyHostToDevice);
    hipMemcpy(d_vdW, vdW, size_vdW, hipMemcpyHostToDevice);
    hipMemcpy(d_Ele,    Ele,    size_atom,   hipMemcpyHostToDevice);
    hipMemcpy(d_q_S_ref_dS,  q_S_ref_dS, 3 * size_q,      hipMemcpyHostToDevice);
    hipMemcpy(d_WK,     WK,     size_WK,     hipMemcpyHostToDevice);
    hipMemcpy(d_c2,     c2,     size_c2,     hipMemcpyHostToDevice);

    float sigma2 = 1.0;
    float alpha = 1.0;
     
     
    dist_calc<<<1024, 1024>>>(d_coord, d_dx, d_dy, d_dz, d_r2, d_close_flag, num_atom, num_atom2); 
    //printf("Done dist_calc\n");
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if(error!=hipSuccess)
    {
       fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
       exit(-1);
    }
    pre_scan_close<<<2048,1024>>>(d_close_flag, d_close_num, d_close_idx, num_atom2);
    //printf("Done pre_scan_close\n");
    hipDeviceSynchronize();
    error = hipGetLastError();
    if(error!=hipSuccess)
    {
       fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
       exit(-1);
    }
    surf_calc<<<1024,512>>>(d_coord, d_Ele, d_r2, d_close_num, d_close_idx, d_vdW, num_atom, num_atom2, num_raster, sol_s, d_V, d_surf, d_surf_grad, offset);
    //printf("Done surf_calc\n");
    hipDeviceSynchronize();
    error = hipGetLastError();
    if(error!=hipSuccess)
    {
       fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
       exit(-1);
    }
    sum_V<<<1,1024>>>(d_V, num_atom, num_atom2, d_Ele, d_vdW);
    //printf("Done sum_V\n");
    hipDeviceSynchronize();
    error = hipGetLastError();
    if(error!=hipSuccess)
    {
       fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
       exit(-1);
    }
    FF_calc<<<320, 32>>>(d_q_S_ref_dS, d_WK, d_vdW, num_q, num_ele, c1, r_m, d_FF_table); 
    //printf("Done FF_calc\n");
    hipDeviceSynchronize();
    error = hipGetLastError();
    if(error!=hipSuccess)
    {
       fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
       exit(-1);
    }
    scat_calc<<<320, 1024>>>(d_coord,  d_Force,   d_Ele,     d_WK,     d_q_S_ref_dS, 
                             d_S_calc, num_atom,  num_q,     num_ele,  d_Aq, 
                             alpha,    k_chi,     sigma2,    d_f_ptxc, d_f_ptyc, 
                             d_f_ptzc, d_S_calcc, num_atom2, num_q2,   d_vdW,
                             d_c2,        d_V,       r_m,      d_FF_table, d_surf_grad);
    //printf("Done scat_calc\n");
    hipDeviceSynchronize();
    error = hipGetLastError();
    if(error!=hipSuccess)
    {
       fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
       exit(-1);
    }
    hipMemcpyAsync(S_calc, d_S_calc, size_q,     hipMemcpyDeviceToHost);
    force_calc<<<1024, 512>>>(d_Force, num_atom, num_q, d_f_ptxc, d_f_ptyc, 
                              d_f_ptzc, num_atom2, num_q2, d_Ele);
    //printf("Done force_calc\n");

    hipDeviceSynchronize();
    error = hipGetLastError();
    if(error!=hipSuccess)
    {
       fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
       exit(-1);
    }


    hipMemcpy(Force,  d_Force,  size_coord, hipMemcpyDeviceToHost);

 

    float chi = 0.0;
    float chi2 = 0.0;
    float chi_ref = 0.0;
    for (int ii = 0; ii < num_q; ii++) {
        chi = q_S_ref_dS[ii+2*num_q] - (S_calc[ii] - q_S_ref_dS[ii+num_q]);
        chi2 += chi * chi;
        chi_ref+= q_S_ref_dS[ii+2*num_q] * q_S_ref_dS[ii+2*num_q];
        //printf("%.3f\n", S_calc[ii]);
    }
    printf("chi square is %.5e ( %.3f \% )\n", chi2, chi2 / chi_ref * 100);
    /*for (int ii = 0; ii < 3 * num_atom; ii++) {
        printf("%.8f ", Force[ii]);
        if ((ii+1) % 3 == 0) printf("\n");
    }*/

 
    hipFree(d_Aq);
    hipFree(d_coord); hipFree(d_Force); 
    hipFree(d_FF); 
    hipFree(d_Ele); 
    hipFree(d_q_S_ref_dS); 
    hipFree(d_S_calc); 
    hipFree(d_f_ptxc); hipFree(d_f_ptyc); hipFree(d_f_ptzc);
    hipFree(d_S_calcc); hipFree(d_WK);
    hipFree(d_dx); hipFree(d_dy); hipFree(d_dz);
    hipFree(d_V); 
    hipFree(d_r2);
    hipFree(d_close_flag); hipFree(d_close_num); hipFree(d_close_idx);
    hipFree(d_vdW);
    hipFree(d_FF_table);
    hipFree(d_surf); hipFree(d_surf_grad);
    hipFree(d_c2);
    free(S_calc);

}
