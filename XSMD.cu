
#include <stdio.h>
#include <math.h>
#include "kernel.cu"
#include "XSMD.hh"
#include "param.hh"

void XSMD_calc (double *coord, double *Force) {
    //int *Ele, double *FF, double *q, double *S_ref, double *dS, double *S_calc, int num_atom, int num_q, int num_ele, double k_chi)
    double *d_Aq, *d_coord, *d_Force, *d_FF, *d_q;
    int *d_Ele;
    double *d_S_ref, *d_dS, *d_S_calc;
    double *S_calc;
    double *d_S_calcc, *d_f_ptxc, *d_f_ptyc, *d_f_ptzc;
    double *d_rot_pt, *d_rot;
    int *d_bond_pp;
    //int *a, *d_a; 
    //a = (int *)malloc(sizeof(int)); 
    //hipMalloc((void **)&d_a,sizeof(int));
    //hipMemset(d_a, 0, sizeof(int));
    int size_coord = 3 * num_atom * sizeof(double);
    int size_atom = num_atom * sizeof(int);
    int size_atom2 = num_atom * sizeof(double);
    int size_q = num_q * sizeof(double); 
    int size_FF = num_ele * num_q * sizeof(double);
    int size_qxatom2 = num_q2 * num_atom2 * sizeof(double); // check if overflow
    int size_bond_pp = 3 * num_pp * sizeof(int);
    int size_rot = num_pp * sizeof(int);
    int size_rotxatom2 = num_pp * num_atom2 * sizeof(double);
    // Initialize Force array
    //Force = (double *)malloc(size_coord);
    for (int ii = 0; ii<3*num_atom; ii++) {
        Force[ii] = 0.0;
    }

    S_calc = (double *)malloc(size_q);
    //for (int ii = 0; ii < num_q; ii++) {
    //    S_calc[ii] = 0.0;
    //}

    hipMalloc((void **)&d_Aq,     size_q);
    hipMemset(d_Aq, 0.0, size_q);
    hipMalloc((void **)&d_coord,  size_coord); // 40 KB
    hipMalloc((void **)&d_Force,  size_coord); // 40 KB
    hipMemset(d_Force, 0.0, size_coord);
    hipMalloc((void **)&d_FF,     size_FF);  // 10 KB ?
    hipMalloc((void **)&d_q,      size_q);  
    hipMalloc((void **)&d_Ele,    size_atom);
    hipMalloc((void **)&d_S_ref,  size_q);
    hipMalloc((void **)&d_dS,     size_q);
    hipMalloc((void **)&d_S_calc, size_q); // Will be computed on GPU
    hipMemset(d_S_calc, 0.0, size_q);
    hipMalloc((void **)&d_f_ptxc, size_qxatom2);
    hipMemset(d_f_ptxc,0.0, size_qxatom2);
    hipMalloc((void **)&d_f_ptyc, size_qxatom2);
    hipMemset(d_f_ptyc,0.0, size_qxatom2);   
    hipMalloc((void **)&d_f_ptzc, size_qxatom2);
    hipMemset(d_f_ptzc,0.0, size_qxatom2);
    hipMalloc((void **)&d_S_calcc, size_qxatom2);
    hipMemset(d_S_calcc,0.0, size_qxatom2);   
    hipMalloc((void **)&d_rot, size_rot);
    hipMemset(d_rot,0.0, size_rot);
    hipMalloc((void **)&d_rot_pt, size_rotxatom2);
    hipMemset(d_rot_pt,0.0, size_rotxatom2);
    hipMalloc((void **)&d_bond_pp, size_bond_pp);
    hipMemcpy(d_coord, coord, size_coord, hipMemcpyHostToDevice);
    //hipMemcpy(d_Force, Force, size_coord, hipMemcpyHostToDevice);
    hipMemcpy(d_q,      q,      size_q,      hipMemcpyHostToDevice);
    hipMemcpy(d_FF,     FF,     size_FF,     hipMemcpyHostToDevice);
    hipMemcpy(d_Ele,    Ele,    size_atom,   hipMemcpyHostToDevice);
    hipMemcpy(d_S_ref,  S_ref,  size_q,      hipMemcpyHostToDevice);
    hipMemcpy(d_dS,     dS,     size_q,      hipMemcpyHostToDevice);
    hipMemcpy(d_bond_pp,bond_pp,size_bond_pp,hipMemcpyHostToDevice);


    //k_chi = 5e-10;
    double sigma2 = 1.0;
    double alpha = 1.0;
    //printf("About to start force_calc...\n");
    scat_calc<<<512, 128>>>(d_coord, d_Force, d_Ele, d_FF, d_q, d_S_ref, d_dS, d_S_calc, num_atom, num_q, num_ele, d_Aq, alpha, k_chi, sigma2, d_f_ptxc, d_f_ptyc, d_f_ptzc, d_S_calcc, num_atom2, num_q2);
    //printf("force_calc finished! \n");
    force_calc<<<128, 512>>>(d_Force, d_q, num_atom, num_q, d_f_ptxc, d_f_ptyc, d_f_ptzc, num_atom2, num_q2);
    
    force_proj<<<32, 128>>>(d_coord, d_Force, d_rot, d_rot_pt, d_bond_pp, num_pp, num_atom, num_atom2);
    //hipMemcpy(rot,    d_rot,    size_rot,   hipMemcpyDeviceToHost);
    pp_assign<<<1, 128>>>(d_coord, d_Force, d_rot, d_bond_pp, num_pp, num_atom);
    hipMemcpy(S_calc, d_S_calc, size_q,     hipMemcpyDeviceToHost);
    hipMemcpy(Force,  d_Force,  size_coord, hipMemcpyDeviceToHost);

 

    //hipMemcpy(a,      d_a,      sizeof(int),hipMemcpyDeviceToHost);
    double chi = 0.0;
    double chi2 = 0.0;
    for (int ii = 0; ii < num_q; ii++) {
        chi = dS[ii] - (S_calc[ii] - S_ref[ii]);
        chi2 += chi * chi;
    }
    /*for (int ii = 0; ii < 3 * num_atom; ii++) {
        printf("%.8f ", Force[ii]);
        if ((ii+1) % 3 == 0) printf("\n");
    }*/
    printf("chi square is %.5e ( %.3f \% )\n",chi2, chi2 / 7.80177e+10 * 100);
    /*for (int ii = 0; ii < 1; ii++) {
        printf("S0 = %.5e \n", S_calc[ii]);
    }*/

    hipFree(d_coord); hipFree(d_Force); hipFree(d_q);
    hipFree(d_Ele); hipFree(d_FF); hipFree(d_S_ref); 
    hipFree(d_dS); hipFree(d_S_calc); hipFree(d_Aq);
    hipFree(d_f_ptxc); hipFree(d_f_ptyc); hipFree(d_f_ptzc);
    hipFree(d_S_calcc);

    hipFree(d_rot); hipFree(d_rot_pt); cudafree(d_bond_pp)
    //hipFree(d_a); free(a);
    free(S_calc);


}
