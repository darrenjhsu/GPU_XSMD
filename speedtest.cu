#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "kernel.cu"
#include "speedtest.hh"
#include "param.hh"
#include "coord_ref.hh"
//#include "raster8.hh"

int main () {
    //int *Ele, float *FF, float *q, float *S_ref, float *dS, float *S_calc, int num_atom, int num_q, int num_ele, float k_chi)
    //for (int ii = 0; ii < num_atom; ii ++) printf("%.3f, %.3f, %.3f\n",coord_ref[ii*3],coord_ref[ii*3+1],coord_ref[ii*3+2]) ;
    hipFree(0);
    //hipDeviceSetCacheConfig(hipFuncCachePreferL1); 
    float *d_Aq, *d_coord, *d_Force, *d_FF;
    int *d_Ele;
    float *d_q_S_ref_dS, *d_S_calc;
    float *S_calc;
    float *d_S_calcc, *d_f_ptxc, *d_f_ptyc, *d_f_ptzc;
    float *d_dx, *d_dy, *d_dz;
    float *d_raster, *d_V, *d_r2;
    //float *d_rot_pt, *d_rot;
    float *d_WK;
    float *Force;
    int *d_close_flag, *d_close_num, *d_close_idx;
    float *d_vdW;
    int *close_num, *close_idx;
    float *V;
    float *d_FF_table;
    float *d_surf, *surf;
    float *d_surf_grad, *surf_grad;
    
    //int *d_bond_pp;
    //int *a, *d_a; 
    //a = (int *)malloc(sizeof(int)); 
    //hipMalloc((void **)&d_a,sizeof(int));
    //hipMemset(d_a, 0, sizeof(int));
    int size_coord = 3 * num_atom * sizeof(float);
    int size_atom = num_atom * sizeof(int);
    int size_atom2 = num_atom2 * sizeof(int);
    int size_atom2f = num_atom2 * sizeof(float);
    int size_atomxatom = num_atom * num_atom * sizeof(float);
    int size_atom2xatom2 = num_atom2 * num_atom2 * sizeof(int);
    printf("size_atomxatom = %d. \n", size_atomxatom);
    int size_q = num_q * sizeof(float); 
    int size_FF = num_ele * num_q * sizeof(float);
    int size_qxatom2 = num_q2 * num_atom2 * sizeof(float); // check if overflow
    int size_raster = num_raster * 3 * sizeof(float);
    int size_FF_table = (num_ele+1) * num_q * sizeof(float);
    int size_surf = num_atom * num_raster * 3 * sizeof(float);
    /*int size_bond_pp = 3 * num_pp * sizeof(int);
    int size_rot = num_pp * sizeof(int);
    int size_rotxatom2 = num_pp * num_atom2 * sizeof(float);*/
    int size_WK = 11 * num_ele * sizeof(float);
    int size_vdW = (num_ele+1) * sizeof(float);
    // Initialize Force array
    Force = (float *)malloc(size_coord);
    close_idx = (int *)malloc(size_atom2xatom2);
    close_num = (int *)malloc(size_atom2);
    V = (float *)malloc(size_atom2f);
    surf_grad = (float *)malloc(size_coord);
    /*for (int ii = 0; ii<3*num_atom; ii++) {
        Force[ii] = 0.0;
    }*/
    /*for (int ii = 0; ii < 66; ii ++) {
        printf("CPU: WK element %d is %.3f\n", ii, WK[ii]);
    }*/
    S_calc = (float *)malloc(size_q);
    surf = (float *)malloc(size_surf);
    //for (int ii = 0; ii < num_q; ii++) {
    //    S_calc[ii] = 0.0;
    //}
    /*float free_m,total_m,used_m;
    size_t free_t,total_t;
    hipMemGetInfo(&free_t,&total_t);
    free_m =(uint)free_t/1048576.0 ;
    total_m=(uint)total_t/1048576.0;
    used_m=total_m-free_m;
    printf ( "  mem: free %d .... %f MB mem total %d....%f MB mem used %f MB\n",free_t,free_m,total_t,total_m,used_m);
    */
    hipMalloc((void **)&d_Aq,     size_q);
    hipMemset(d_Aq, 0.0, size_q);
    hipMalloc((void **)&d_coord,  size_coord); // 40 KB
    hipMalloc((void **)&d_Force,  size_coord); // 40 KB
    hipMemset(d_Force, 0.0, size_coord);
    hipMalloc((void **)&d_FF,     size_FF);  // 10 KB ?
    hipMalloc((void **)&d_Ele,    size_atom);
    hipMalloc((void **)&d_q_S_ref_dS, 3 * size_q);
    hipMalloc((void **)&d_S_calc, size_q); // Will be computed on GPU
    hipMemset(d_S_calc, 0.0, size_q);
    hipMalloc((void **)&d_f_ptxc, size_qxatom2);
    hipMemset(d_f_ptxc,0.0, size_qxatom2);
    hipMalloc((void **)&d_f_ptyc, size_qxatom2);
    hipMemset(d_f_ptyc,0.0, size_qxatom2);   
    hipMalloc((void **)&d_f_ptzc, size_qxatom2);
    hipMemset(d_f_ptzc,0.0, size_qxatom2);
    hipMalloc((void **)&d_S_calcc, size_qxatom2);
    hipMemset(d_S_calcc,0.0, size_qxatom2);
    //hipMalloc((void **)&d_raster, size_raster);
    hipMalloc((void **)&d_V, size_atom2f);
    hipMalloc((void **)&d_dx, size_atomxatom);
    hipMalloc((void **)&d_dy, size_atomxatom);
    hipMalloc((void **)&d_dz, size_atomxatom);
    hipMalloc((void **)&d_r2, size_atomxatom);
    //hipMemcpy(d_raster, raster, size_raster, hipMemcpyHostToDevice); 
    hipMalloc((void **)&d_close_flag, size_atom2xatom2);
    hipMemset(d_close_flag, 0, size_qxatom2);
    hipMalloc((void **)&d_close_num, size_atom2);
    hipMemset(d_close_num, 0, size_atom2);
    hipMalloc((void **)&d_close_idx, size_atom2xatom2);
    hipMemset(d_close_idx, 0, size_atom2xatom2);
    hipMalloc((void **)&d_vdW, size_vdW);
    hipMalloc((void **)&d_FF_table, size_FF_table);
    hipMemcpy(d_vdW, vdW, size_vdW, hipMemcpyHostToDevice);
    /*hipMalloc((void **)&d_rot, size_rot);
    hipMemset(d_rot,0.0, size_rot);
    hipMalloc((void **)&d_rot_pt, size_rotxatom2);
    hipMemset(d_rot_pt,0.0, size_rotxatom2);
    hipMalloc((void **)&d_bond_pp, size_bond_pp);*/
    hipMalloc((void **)&d_WK, size_WK);
    hipMalloc((void **)&d_surf, size_surf);
    hipMemset(d_surf, 0.0, size_surf);
    hipMalloc((void **)&d_surf_grad, size_coord);
    hipMemset(d_surf_grad, 0.0, size_coord);
    hipMemcpy(d_coord, coord_ref, size_coord,    hipMemcpyHostToDevice);
    //hipMemcpy(d_Force, Force, size_coord, hipMemcpyHostToDevice);
    //hipMemcpy(d_q,      q,      size_q,      hipMemcpyHostToDevice);
    //hipMemcpy(d_FF,     FF,     size_FF,     hipMemcpyHostToDevice);
    hipMemcpy(d_Ele,    Ele,    size_atom,   hipMemcpyHostToDevice);
    hipMemcpy(d_q_S_ref_dS,  q_S_ref_dS, 3 * size_q,      hipMemcpyHostToDevice);
    //hipMemcpy(d_dS,     dS,     size_q,      hipMemcpyHostToDevice);
    //hipMemcpy(d_bond_pp,bond_pp,size_bond_pp,hipMemcpyHostToDevice);
    hipMemcpy(d_WK,     WK,     size_WK,     hipMemcpyHostToDevice);
    //printf("Finished copying.\n");

    //k_chi = 5e-10;
    float sigma2 = 1.0;
    float alpha = 1.0;
     
    //printf("About to start force_calc...\n");
    //scat_calc<<<512, 128>>>(d_coord, d_Force, d_Ele, d_FF, d_q, d_S_ref, d_dS, d_S_calc, num_atom, num_q, num_ele, d_Aq, alpha, k_chi, sigma2, d_f_ptxc, d_f_ptyc, d_f_ptzc, d_S_calcc, num_atom2, num_q2);
    dist_calc<<<1024, 1024>>>(d_coord, d_dx, d_dy, d_dz, d_r2, d_close_flag, num_atom, num_atom2); 
    pre_scan_close<<<2048,1024>>>(d_close_flag, d_close_num, d_close_idx, num_atom2);
    hipMemcpy(close_num, d_close_num, size_atom2, hipMemcpyDeviceToHost);
    //hipMemcpy(close_idx, d_close_idx, size_atom2xatom2, hipMemcpyDeviceToHost);
    //surf_calc<<<1024,512>>>(d_coord, d_Ele, d_r2, d_close_num, d_close_idx, d_vdW, num_atom, num_atom2, num_raster, sol_s, d_V, d_surf, d_surf_grad, offset);
    surf_calc<<<1024,512>>>(d_coord, d_Ele, d_r2, d_close_num, d_close_idx, d_vdW, num_atom, num_atom2, num_raster, sol_s, d_V, d_surf, d_surf_grad, offset);
    //hipMemcpy(surf_grad, d_surf_grad, size_coord, hipMemcpyDeviceToHost);
    sum_V<<<1,1024>>>(d_V, num_atom, num_atom2, d_Ele, d_vdW);
    hipMemcpy(V, d_V, size_atom2f, hipMemcpyDeviceToHost);

    // Print surf info
    /*
    for (int i = 0; i < num_atom; i++) {
        printf("%3d atoms are close to atom %4d, %.6f of surf being exposed.\n", close_num[i], i, V[i]);
        //for (int j = 0; j < close_num[i]; j++) {
        //for (int j = 0; j < 30; j++) {
            printf("%4d, ", close_idx[i*num_atom2+j]);
        }
        //printf("\n");
    }
    */
    //border_scat<<<1024, 1024>>>(d_coord, d_Ele, d_r2, d_raster, d_V, num_atom, num_atom2, num_raster, num_raster2); 
    //V_calc<<<1, 1024>>>(d_V, num_atom2);
    FF_calc<<<320, 32>>>(d_q_S_ref_dS, d_WK, d_vdW, num_q, num_ele, c1, r_m, d_FF_table); 
    scat_calc<<<320, 1024>>>(d_coord,  d_Force,   d_Ele,     d_WK,     d_q_S_ref_dS, 
                             d_S_calc, num_atom,  num_q,     num_ele,  d_Aq, 
                             //d_S_calc, num_atom,  num_q,     num_ele, 
                             alpha,    k_chi,     sigma2,    d_f_ptxc, d_f_ptyc, 
                             d_f_ptzc, d_S_calcc, num_atom2, num_q2,   d_vdW,
                             c2,       d_V,       r_m,      d_FF_table, 
                             d_surf_grad);
    //printf("force_calc finished! \n");
    //printf("%d \n",hipDeviceSynchronize());
    // hipDeviceSynchronize();
    hipMemcpy(S_calc, d_S_calc, size_q,     hipMemcpyDeviceToHost);
    hipMemcpy(surf,   d_surf,   size_surf,  hipMemcpyDeviceToHost);
    force_calc<<<1024, 512>>>(d_Force, num_atom, num_q, d_f_ptxc, d_f_ptyc, 
                              d_f_ptzc, num_atom2, num_q2, d_Ele);
    
    //printf("%d \n",hipDeviceSynchronize());
    //force_proj<<<32, 128>>>(d_coord, d_Force, d_rot, d_rot_pt, d_bond_pp, num_pp, num_atom, num_atom2);
    //hipMemcpy(rot,    d_rot,    size_rot,   hipMemcpyDeviceToHost);
    //pp_assign<<<1, 128>>>(d_coord, d_Force, d_rot, d_bond_pp, num_pp, num_atom);

    hipMemcpy(Force,  d_Force,  size_coord, hipMemcpyDeviceToHost);


    //hipMemcpy(a,      d_a,      sizeof(int),hipMemcpyDeviceToHost);
    float chi = 0.0;
    float chi2 = 0.0;
    float chi_ref = 0.0;
    for (int ii = 0; ii < num_q; ii++) {
        chi = q_S_ref_dS[ii+2*num_q] - (S_calc[ii] - q_S_ref_dS[ii+num_q]);
        printf("q = %.3f: chi is: %.3f, dS is: %.3f, S_calc is: %.3f, S_ref is: %.3f\n", q_S_ref_dS[ii], chi, q_S_ref_dS[ii+2*num_q], S_calc[ii], q_S_ref_dS[ii+num_q]); 
        chi2 += chi * chi;
        chi_ref+= q_S_ref_dS[ii+2*num_q] * q_S_ref_dS[ii+2*num_q];
    }
    for (int ii = 0; ii < 3 * num_atom; ii++) {
        printf("%d: %.8f ", ii/3, Force[ii]);
        if ((ii+1) % 3 == 0) printf("\n");
    }
    for (int ii = 0; ii < num_atom; ii++) {
        printf("grad: %4d: %7.4f %7.4f %7.4f \n", ii, surf_grad[3*ii], surf_grad[3*ii+1], surf_grad[3*ii+2]);
    }
    printf("chi square is %.5e ( %.3f \% )\n", chi2, chi2 / chi_ref * 100);
    /*for (int ii = 0; ii < 1; ii++) {
        printf("S0 = %.5e \n", S_calc[ii]);
    }*/

    // Print surface points
    /*
            printf("CRYST1    0.000    0.000    0.000  90.00  90.00  90.00 P 1           1\n");
    int idx = 0;
    for (int ii = 0; ii < num_atom * num_raster; ii++) {
        if (surf[3*ii] != 0) {
            printf("ATOM  %5d  XXX XXX P   1     %7.3f %7.3f %7.3f  0.00  0.00      P1\n", idx, surf[3*ii], surf[3*ii+1], surf[3*ii+2]);
            idx++;
        }
    }
    */
    hipFree(d_coord); hipFree(d_Force); //hipFree(d_q);
    hipFree(d_Ele); hipFree(d_FF); 
    hipFree(d_q_S_ref_dS); 
    // hipFree(d_dS);
    hipFree(d_S_calc); hipFree(d_Aq);
    hipFree(d_f_ptxc); hipFree(d_f_ptyc); hipFree(d_f_ptzc);
    hipFree(d_S_calcc); hipFree(d_WK);
    hipFree(d_dx); hipFree(d_dy); hipFree(d_dz);
    //hipFree(d_raster); hipFree(d_V); 
    hipFree(d_r2);
    hipFree(d_close_flag); hipFree(d_close_num); hipFree(d_close_idx);
    hipFree(d_vdW);
    hipFree(d_surf_grad);
    //hipFree(d_rot); hipFree(d_rot_pt); hipFree(d_bond_pp);
    //hipFree(d_a); free(a);
    free(S_calc); free(close_num); free(close_idx); free(surf_grad);
    //printf("So the fault is at NAMD?\n");

    return 0;
}
