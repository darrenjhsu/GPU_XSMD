#include <stdio.h>
#include <math.h>
#include "kernel.cu"
#include "speedtest.hh"
#include "param.hh"
#include "coord_ref.hh"


int main () {
    //int *Ele, float *FF, float *q, float *S_ref, float *dS, float *S_calc, int num_atom, int num_q, int num_ele, float k_chi)
    //for (int ii = 0; ii < num_atom; ii ++) printf("%.3f, %.3f, %.3f\n",coord_ref[ii*3],coord_ref[ii*3+1],coord_ref[ii*3+2]) ;
    float *d_Aq, *d_coord, *d_Force, *d_FF;
    int *d_Ele;
    float *d_q_S_ref_dS, *d_S_calc;
    float *S_calc;
    float *d_S_calcc, *d_f_ptxc, *d_f_ptyc, *d_f_ptzc;
    //float *d_rot_pt, *d_rot;
    float *d_WK;
    float *Force;
    //int *d_bond_pp;
    //int *a, *d_a; 
    //a = (int *)malloc(sizeof(int)); 
    //hipMalloc((void **)&d_a,sizeof(int));
    //hipMemset(d_a, 0, sizeof(int));
    int size_coord = 3 * num_atom * sizeof(float);
    int size_atom = num_atom * sizeof(int);
    int size_atom2 = num_atom * sizeof(float);
    int size_q = num_q * sizeof(float); 
    int size_FF = num_ele * num_q * sizeof(float);
    int size_qxatom2 = num_q2 * num_atom2 * sizeof(float); // check if overflow
    //printf("size_qxatom2 = %d. \n", size_qxatom2);
    /*int size_bond_pp = 3 * num_pp * sizeof(int);
    int size_rot = num_pp * sizeof(int);
    int size_rotxatom2 = num_pp * num_atom2 * sizeof(float);*/
    int size_WK = 11 * num_ele * sizeof(float);
    // Initialize Force array
    Force = (float *)malloc(size_coord);
    /*for (int ii = 0; ii<3*num_atom; ii++) {
        Force[ii] = 0.0;
    }*/
    /*for (int ii = 0; ii < 66; ii ++) {
        printf("CPU: WK element %d is %.3f\n", ii, WK[ii]);
    }*/
    S_calc = (float *)malloc(size_q);
    //for (int ii = 0; ii < num_q; ii++) {
    //    S_calc[ii] = 0.0;
    //}
    
    hipMalloc((void **)&d_Aq,     size_q);
    hipMemset(d_Aq, 0.0, size_q);
    hipMalloc((void **)&d_coord,  size_coord); // 40 KB
    hipMalloc((void **)&d_Force,  size_coord); // 40 KB
    hipMemset(d_Force, 0.0, size_coord);
    hipMalloc((void **)&d_FF,     size_FF);  // 10 KB ?
    hipMalloc((void **)&d_Ele,    size_atom);
    hipMalloc((void **)&d_q_S_ref_dS, 3 * size_q);
    hipMalloc((void **)&d_S_calc, size_q); // Will be computed on GPU
    hipMemset(d_S_calc, 0.0, size_q);
    hipMalloc((void **)&d_f_ptxc, size_qxatom2);
    hipMemset(d_f_ptxc,0.0, size_qxatom2);
    hipMalloc((void **)&d_f_ptyc, size_qxatom2);
    hipMemset(d_f_ptyc,0.0, size_qxatom2);   
    hipMalloc((void **)&d_f_ptzc, size_qxatom2);
    hipMemset(d_f_ptzc,0.0, size_qxatom2);
    hipMalloc((void **)&d_S_calcc, size_qxatom2);
    hipMemset(d_S_calcc,0.0, size_qxatom2);
    /*hipMalloc((void **)&d_rot, size_rot);
    hipMemset(d_rot,0.0, size_rot);
    hipMalloc((void **)&d_rot_pt, size_rotxatom2);
    hipMemset(d_rot_pt,0.0, size_rotxatom2);
    hipMalloc((void **)&d_bond_pp, size_bond_pp);*/
    hipMalloc((void **)&d_WK, size_WK);
    hipMemcpy(d_coord, coord_ref, size_coord,    hipMemcpyHostToDevice);
    //hipMemcpy(d_Force, Force, size_coord, hipMemcpyHostToDevice);
    //hipMemcpy(d_q,      q,      size_q,      hipMemcpyHostToDevice);
    //hipMemcpy(d_FF,     FF,     size_FF,     hipMemcpyHostToDevice);
    hipMemcpy(d_Ele,    Ele,    size_atom,   hipMemcpyHostToDevice);
    hipMemcpy(d_q_S_ref_dS,  q_S_ref_dS, 3* size_q,      hipMemcpyHostToDevice);
    //hipMemcpy(d_dS,     dS,     size_q,      hipMemcpyHostToDevice);
    //hipMemcpy(d_bond_pp,bond_pp,size_bond_pp,hipMemcpyHostToDevice);
    hipMemcpy(d_WK,     WK,     size_WK,     hipMemcpyHostToDevice);
    //printf("Finished copying.\n");

    //k_chi = 5e-10;
    float sigma2 = 1.0;
    float alpha = 1.0;
    //printf("About to start force_calc...\n");
    //scat_calc<<<512, 128>>>(d_coord, d_Force, d_Ele, d_FF, d_q, d_S_ref, d_dS, d_S_calc, num_atom, num_q, num_ele, d_Aq, alpha, k_chi, sigma2, d_f_ptxc, d_f_ptyc, d_f_ptzc, d_S_calcc, num_atom2, num_q2);
    scat_calc<<<320, 1024>>>(d_coord, d_Force, d_Ele, d_WK, d_q_S_ref_dS, d_S_calc, num_atom, num_q, num_ele, d_Aq, alpha, k_chi, sigma2, d_f_ptxc, d_f_ptyc, d_f_ptzc, d_S_calcc, num_atom2, num_q2);
    //printf("force_calc finished! \n");
    //printf("%d \n",hipDeviceSynchronize());
    hipMemcpy(S_calc, d_S_calc, size_q,     hipMemcpyDeviceToHost);
    force_calc<<<1024, 512>>>(d_Force, num_atom, num_q, d_f_ptxc, d_f_ptyc, d_f_ptzc, num_atom2, num_q2);
    
    //printf("%d \n",hipDeviceSynchronize());
    //force_proj<<<32, 128>>>(d_coord, d_Force, d_rot, d_rot_pt, d_bond_pp, num_pp, num_atom, num_atom2);
    //hipMemcpy(rot,    d_rot,    size_rot,   hipMemcpyDeviceToHost);
    //pp_assign<<<1, 128>>>(d_coord, d_Force, d_rot, d_bond_pp, num_pp, num_atom);
    hipMemcpy(Force,  d_Force,  size_coord, hipMemcpyDeviceToHost);

 

    //hipMemcpy(a,      d_a,      sizeof(int),hipMemcpyDeviceToHost);
    float chi = 0.0;
    float chi2 = 0.0;
    float chi_ref = 0.0;
    for (int ii = 0; ii < num_q; ii++) {
        chi = q_S_ref_dS[ii+2*num_q] - (S_calc[ii] - q_S_ref_dS[ii+num_q]);
        //printf("%d: chi is: %.3f, dS is: %.3f, S_calc is: %.3f, S_ref is: %.3f\n", ii, chi, dS[ii], S_calc[ii], S_ref[ii]); 
        chi2 += chi * chi;
        chi_ref+= q_S_ref_dS[ii+2*num_q] * q_S_ref_dS[ii+2*num_q];
    }
    /*for (int ii = 0; ii < 3 * num_atom; ii++) {
        printf("%.8f ", Force[ii]);
        if ((ii+1) % 3 == 0) printf("\n");
    }*/
    printf("chi square is %.5e ( %.3f \% )\n", chi2, chi2 / chi_ref * 100);
    /*for (int ii = 0; ii < 1; ii++) {
        printf("S0 = %.5e \n", S_calc[ii]);
    }*/

    hipFree(d_coord); hipFree(d_Force); //hipFree(d_q);
    hipFree(d_Ele); hipFree(d_FF); 
    hipFree(d_q_S_ref_dS); 
    // hipFree(d_dS);
    hipFree(d_S_calc); hipFree(d_Aq);
    hipFree(d_f_ptxc); hipFree(d_f_ptyc); hipFree(d_f_ptzc);
    hipFree(d_S_calcc); hipFree(d_WK);

    //hipFree(d_rot); hipFree(d_rot_pt); hipFree(d_bond_pp);
    //hipFree(d_a); free(a);
    free(S_calc);
    //printf("So the fault is at NAMD?\n");

    return 0;
}
