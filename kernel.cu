#include "hip/hip_runtime.h"

#include <stdio.h>
#include <math.h>
#include <time.h>
#include "param.hh"
#include "WaasKirf.hh"
#define PI 3.14159265359


/*__device__ float dot (float a1, float a2, float a3, float b1, float b2, float b3) {
    return (a1 * b1 + a2 * b2 + a3 * b3);
}

__device__ float cross2 (float a2, float a3, float b2, float b3) {
    return (a2 * b3 - a3 * b2);
}*/
 
//__global__ void scat_calc (float *coord, float *Force, int *Ele, float *FF, float *q, float *S_ref, float *dS, float *S_calc, int num_atom, int num_q, int num_ele, float *Aq, float alpha, float k_chi, float sigma2, float *f_ptxc, float *f_ptyc, float *f_ptzc, float *S_calcc, int num_atom2, int num_q2) {
/*__global__ void build_cxsxdx (float *cxsxdx_table, float *sx_table, float *qr_vec, float qr_max, float qr_step, int num_bin) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    if (idx >= num_bin) return;
    if (blockIdx.x == 0 && threadIdx.x == 0) printf("qr_max = %.3f, qr_step = %.3f, num_bin = %d. \n", qr_max, qr_step, num_bin);
    for (int ii = idx; ii < num_bin; ii += stride) {
        //Initialize table and qr_vec
        if (ii == 0) {
            qr_vec[ii] = 0.0;
            cxsxdx_table[ii] = 0.0;
            sx_table[ii] = 0.0;
        }
        qr_vec[ii] = (float)ii * qr_step;
        cxsxdx_table[ii] = cos(qr_vec[ii]) - sin(qr_vec[ii]) / qr_vec[ii];
        sx_table[ii] = sin(qr_vec[ii]);
    }

}
*/
__global__ void scat_calc (float *coord, float *Force, int *Ele, float *WK, float *q_S_ref_dS, float *S_calc, int num_atom, int num_q, int num_ele, float *Aq, float alpha, float k_chi, float sigma2, float *f_ptxc, float *f_ptyc, float *f_ptzc, float *S_calcc, int num_atom2, int num_q2) {
    __shared__ float q_pt, q_WK;
    __shared__ float FF_pt[6];
    __shared__ float WK_s[66];
    __shared__ float S_calccs[1024];
    __shared__ float f_ptxcs[1024];
    __shared__ float f_ptycs[1024];
    __shared__ float f_ptzcs[1024];
    //float FF_pt[6]; 
    if (blockIdx.x >= num_q) return; // out of q range
    if (threadIdx.x >= num_atom) return; // out of atom numbers (not happening)
   
    /*if (blockIdx.x == 0) {
        for (int jj = threadIdx.x; jj < num_atom; jj += blockDim.x) {
            f_ptx[jj] = 0.0;
            f_pty[jj] = 0.0;
            f_ptz[jj] = 0.0;
        }
        for (int jj = threadIdx.x; jj < num_q; jj += blockDim.x) {
            Aq[jj] = 0.0;
            S_calc[jj] = 0.0;
        }
    }
    __syncthreads();
    */
    //unsigned int t1, t2, t3;
    //for (int ii = blockIdx.x * blockDim.x + threadIdx.x; ii < num_q; ii += blockDim.x * gridDim.x) {
    for (int ii = blockIdx.x; ii < num_q; ii += gridDim.x) {
          //       0 - 512          300          512
        q_pt = q_S_ref_dS[ii];
        //if (threadIdx.x == 0) {
        //    printf("q_pt[%d] = %.3f. \n", ii, q_pt);
        //}
        q_WK = q_pt / 4.0 / PI;
        // Put FF coeff to shared memory
        //if (threadIdx.x == 0)
        for (int jj = threadIdx.x; jj < 11 * num_ele; jj +=blockDim.x) {
            WK_s[jj] = WK[jj];
        }
        __syncthreads();
        /*if (blockIdx.x == 0 && threadIdx.x == 0) {
            for (int jj = 0; jj < 66; jj ++) {
                printf("WK %d before is %.3f. \n", jj, WK[jj]);
                printf("WK coeff %d is %.3f. \n", jj, WK_s[jj]);
            }
        }
        __syncthreads();*/
        // Calculate Form factor for this block (or q vector)
        for (int jj = threadIdx.x; jj < num_ele; jj += blockDim.x) {
            FF_pt[jj] = WK_s[jj*11] * exp(-WK_s[jj*11+6] * q_WK * q_WK) + \
                        WK_s[jj*11+1] * exp(-WK_s[jj*11+7] * q_WK * q_WK) + \
                        WK_s[jj*11+2] * exp(-WK_s[jj*11+8] * q_WK * q_WK) + \
                        WK_s[jj*11+3] * exp(-WK_s[jj*11+9] * q_WK * q_WK) + \
                        WK_s[jj*11+4] * exp(-WK_s[jj*11+10] * q_WK * q_WK) + \
                        WK_s[jj*11+5];
            //if (ii == 0) printf("FF for elem %d at q = 0 is %.3f.\n", jj, FF_pt[jj]);
        }
        __syncthreads();
        // Calculate scattering for Aq
        //for (int jj = blockIdx.y * blockDim.y + threadIdx.y; jj < num_atom; jj += blockIdx.y * gridDim.y) {
        for (int jj = threadIdx.x; jj < num_atom; jj += blockDim.x) {
              //       0 - 1023          1749            1024
            int idx = jj % blockDim.x;
            // if (jj==1) printf("idx is %d. \n",idx);
            S_calccs[idx] = 0.0; f_ptxcs[idx] = 0.0; f_ptycs[idx] = 0.0; f_ptzcs[idx] = 0.0;
            float atom1x = coord[3*jj+0];
            float atom1y = coord[3*jj+1];
            float atom1z = coord[3*jj+2];
            int atom1t = Ele[jj]; // atom1 element type
            float atom1FF = FF_pt[atom1t]; // atom1 form factor at q // 6 ms
            //float atom1FF = FF[ii*num_ele+atom1t]; // atom1 form factor at q
            for (int kk = 0; kk < num_atom; kk++) {
                int atom2t = Ele[kk]; // 6 ms
                if (q_pt == 0.0) {
                    //S_calcc[ii*num_atom2+jj] += atom1FF * FF[ii*num_ele+atom2t];
                    //S_calcc[ii*num_atom2+jj] += atom1FF * FF_pt[atom2t];
                    S_calccs[idx] += atom1FF * FF_pt[atom2t]; // 6.2 ms
                    //*a = 1;
                } else if (kk == jj) {
                    //S_calcc[ii*num_atom2+jj] += atom1FF * FF[ii*num_ele+atom2t];
                    //S_calcc[ii*num_atom2+jj] += atom1FF * FF_pt[atom2t];
                    S_calccs[idx] += atom1FF * FF_pt[atom2t]; // 7.6 ms
                } else {
                    //if (ii==0&&jj==0&&kk==0) t1 = clock();
                    float dx = coord[3*kk+0] - atom1x;
                    float dy = coord[3*kk+1] - atom1y;
                    float dz = coord[3*kk+2] - atom1z; // 7.6 ms
                    float r = sqrt(dx*dx+dy*dy+dz*dz); // 7.6 ms
                    //if (ii==1&&jj==0&&kk==1) printf("Distance btw jj = 0 and kk = 1 is sqrt (%.3f^2 + %.3f^2 + %.3f^2) = %.3f. \n", dx, dy, dz, r);
                    //int r_bin = (int)(q_pt * r / qr_step);
                    float sqr = sin(q_pt * r); // 22 ms
                    float prefac = atom1FF * FF_pt[atom2t] * (cos(q_pt * r) - sqr / q_pt / r) / r / r; //27 ms
                    //float prefac = atom1FF * FF_pt[atom2t] * cxsxdx_table[r_bin] / r / r;
                    //float prefac = atom1FF * FF[ii*num_ele+atom2t] * (cos(q_pt * r) - sin(q_pt * r) / q_pt / r) / r / r;
                    //S_calcc[ii*num_atom2+jj] += atom1FF * FF[ii*num_ele+atom2t] * sin(q_pt * r) / q_pt / r;
                    //S_calcc[ii*num_atom2+jj] += atom1FF * FF_pt[atom2t] * sqr / q_pt / r;
                    S_calccs[idx] += atom1FF * FF_pt[atom2t] * sqr / q_pt / r; // 51 ms
                    //S_calccs[jj] += atom1FF * FF_pt[atom2t] * sx_table[r_bin] / q_pt / r;
                    // f_ptxc[ii*num_atom2+jj] += prefac * dx;
                    // f_ptyc[ii*num_atom2+jj] += prefac * dy;
                    // f_ptzc[ii*num_atom2+jj] += prefac * dz; // 94 ms
                    f_ptxcs[idx] += prefac * dx;
                    f_ptycs[idx] += prefac * dy; 
                    f_ptzcs[idx] += prefac * dz; // 94 -> 90 ms.
                    //if (ii==0&&jj==0&&kk==num_atom-1) t2 = clock();
                }
            }
            //if (ii==0&&jj==1024) t2=clock();
            S_calcc[ii*num_atom2+jj] = S_calccs[idx];
            //if (ii==0&&jj>0&&jj<10) printf("S_calccs[jj = %d] = %f. \n",jj,S_calccs[idx]);
            f_ptxc[ii*num_atom2+jj] = f_ptxcs[idx];
            f_ptyc[ii*num_atom2+jj] = f_ptycs[idx];
            f_ptzc[ii*num_atom2+jj] = f_ptzcs[idx];
        }
        
        // S_calc[ii] += S_pt;
        // Tree-like summation of S_calcc to get S_calc
        for (int stride = num_atom2 / 2; stride > 0; stride >>= 1) {
            __syncthreads();
            for(int iAccum = threadIdx.x; iAccum < stride; iAccum += blockDim.x) {
                S_calcc[ii * num_atom2 + iAccum] += S_calcc[ii * num_atom2 + stride + iAccum];
            }
        }
        __syncthreads();
        if (threadIdx.x == 0) {
            S_calc[ii] = S_calcc[ii * num_atom2];
            Aq[ii] = k_chi / 2.0 / sigma2 * ( q_S_ref_dS[ii+2*num_q] - alpha * (S_calc[ii] - q_S_ref_dS[ii+num_q]));
            //printf("S_calc[%d] = %.3f. \n", ii, S_calc[ii]);
        }
        __syncthreads();
        // Multiply f_pt{x,y,z}c(q) by Aq(q) * 8 * alpha * k_chi / sigma2
        for (int jj = threadIdx.x; jj < num_atom; jj += blockDim.x) {
            f_ptxc[ii * num_atom2 + jj] *= Aq[ii] * 4.0 * alpha;
            f_ptyc[ii * num_atom2 + jj] *= Aq[ii] * 4.0 * alpha;
            f_ptzc[ii * num_atom2 + jj] *= Aq[ii] * 4.0 * alpha;
        }
        __syncthreads();
        // Call another device function (block = atom_num, threads = num_q)
        // to column sum f_pt{x,y,z}c for Force[jj] 
        /*for (int jj = threadIdx.x; jj < num_atom; jj += blockDim.x) {
              //       0 - 1023          1749            1024
            //Force[jj] += 8 * alpha * k_chi / sigma2 * Aq[ii];
            Force[jj] = 0.0;
        }*/
    }
    /*if (blockIdx.x == 0 && threadIdx.x == 0) {
        t3 = clock();
        printf("Elapsed time: %.3f s for scat calc and %.3f s for overall \n", (float)(t2-t1) / CLOCKS_PER_SEC, (float)(t3-t1) /CLOCKS_PER_SEC);
    }*/
}


__global__ void force_calc (float *Force, int num_atom, int num_q, float *f_ptxc, float *f_ptyc, float *f_ptzc, int num_atom2, int num_q2) {
    // Do column tree sum of f_ptxc for f_ptx for every atom, then assign threadIdx.x == 0 (3 * num_atoms) to Force. Force is num_atom * 3. 
    //if (threadIdx.x == 0) printf("blockIdx.x = %d\n", blockIdx.x);
    if (blockIdx.x >= num_atom) return;
    for (int ii = blockIdx.x; ii < num_atom; ii += gridDim.x) {
        //printf("BlockIdx = %d \n", ii);
        for (int stride = num_q2 / 2; stride > 0; stride >>= 1) {
            __syncthreads();
            for(int iAccum = threadIdx.x; iAccum < stride; iAccum += blockDim.x) {
                f_ptxc[ii + iAccum * num_atom2] += f_ptxc[ii + iAccum * num_atom2 + stride * num_atom2];
                f_ptyc[ii + iAccum * num_atom2] += f_ptyc[ii + iAccum * num_atom2 + stride * num_atom2];
                f_ptzc[ii + iAccum * num_atom2] += f_ptzc[ii + iAccum * num_atom2 + stride * num_atom2];
            }
        }
        __syncthreads();
        if (threadIdx.x == 0) {
            Force[ii*3    ] = -f_ptxc[ii];
            Force[ii*3 + 1] = -f_ptyc[ii];
            Force[ii*3 + 2] = -f_ptzc[ii];
            //Force[ii*3    ] = 0.0;// -f_ptxc[ii];
            //Force[ii*3 + 1] = 0.0;//-f_ptyc[ii];
            //Force[ii*3 + 2] = 0.0;//-f_ptzc[ii];
        }
        __syncthreads();
    }
}

/*__global__ void force_proj (float *coord, float *Force, float *rot, float *rot_pt, int *bond_pp, int num_pp, int num_atom, int num_atom2) {
    if (blockIdx.x >= num_pp) return;
    if (threadIdx.x >= num_atom) return;
    for (int ii = blockIdx.x; ii < num_pp; ii += gridDim.x) {
        // For each pp bond
        // Calculate normalized torsional vector
        float cp1 = 0.0;
        float cp2 = 0.0;
        float cp3 = 0.0; // Cross product
        int E1, E2, E3; // Atom index of the pp bond
        E1 = bond_pp[3*ii]; E2 = bond_pp[3*ii+1]; E3 = bond_pp[3*ii+2];
        //if (ii == 0) printf("Elements are %d %d and %d \n", E1, E2, E3);
        cp1 = cross2(coord[3*E2+1]-coord[3*E1+1], coord[3*E2+2]-coord[3*E1+2],
                     coord[3*E3+1]-coord[3*E2+1], coord[3*E3+2]-coord[3*E2+2]);
        cp2 = cross2(coord[3*E2+2]-coord[3*E1+2], coord[3*E2+0]-coord[3*E1+0],
                     coord[3*E3+2]-coord[3*E2+2], coord[3*E3+0]-coord[3*E2+0]);
        cp3 = cross2(coord[3*E2+0]-coord[3*E1+0], coord[3*E2+1]-coord[3*E1+1],
                     coord[3*E3+0]-coord[3*E2+0], coord[3*E3+1]-coord[3*E2+1]);
        float r = sqrt(cp1 * cp1 + cp2 * cp2 + cp3 * cp3);
        cp1 /= r;
        cp2 /= r;
        cp3 /= r;

        for (int jj = threadIdx.x; jj < num_atom; jj += blockDim.x) {
            // For each atom
            if (jj == E1 || jj == E2) {
                continue;
            } else {
                rot_pt[ii*num_atom2+jj] += dot(cp1, cp2, cp3, coord[3*jj], coord[3*jj+1], coord[3*jj+2]);
            }
        }
    

    // Perform summation for rot
        for (int stride = num_atom2 / 2; stride > 0; stride >>= 1) {
            __syncthreads();
            for(int iAccum = threadIdx.x; iAccum < stride; iAccum += blockDim.x) {
                rot_pt[ii * num_atom2 + iAccum] += rot_pt[ii * num_atom2 + stride + iAccum];
            }
        }
        __syncthreads();
        if (threadIdx.x == 0) {
            rot[ii] = rot_pt[ii * num_atom2];
        }
        __syncthreads();
    }   

}
*/
/*
__global__ void pp_assign (float *coord, float *Force, float *rot, int *bond_pp, int num_pp, int num_atom) {
    if (threadIdx.x >= num_atom) return;
    for (int ii = threadIdx.x; ii < num_atom; ii += blockDim.x) {
        Force[3*ii] = 0.0;
        Force[3*ii+1] = 0.0;
        Force[3*ii+2] = 0.0;
    }
    __syncthreads();
    if (threadIdx.x == 0) {
        for (int ii = 0; ii < num_atom; ii ++) {
            printf("Force is now %.3f, %.3f, and %.3f. \n", Force[ii], Force[ii+1], Force[ii+2]);
        }
        printf("rot values: \n");
        for (int ii = 0; ii < num_pp; ii++) {
            printf("%.3f \n", rot[ii]*1e3);
        }
    }
    __syncthreads();
    for (int ii = threadIdx.x; ii < num_pp; ii += blockDim.x) {
        float cp1 = 0.0;
        float cp2 = 0.0;
        float cp3 = 0.0; // Cross product
        int E1, E2, E3; // Atom index of the pp bond
        E1 = bond_pp[3*ii]; E2 = bond_pp[3*ii+1]; E3 = bond_pp[3*ii+2];
        //printf("Element 3 is %d. \n", E3);
        cp1 = cross2(coord[3*E2+1]-coord[3*E1+1], coord[3*E2+2]-coord[3*E1+2],
                     coord[3*E3+1]-coord[3*E2+1], coord[3*E3+2]-coord[3*E2+2]);
        cp2 = cross2(coord[3*E2+2]-coord[3*E1+2], coord[3*E2+0]-coord[3*E1+0],
                     coord[3*E3+2]-coord[3*E2+2], coord[3*E3+0]-coord[3*E2+0]);
        cp3 = cross2(coord[3*E2+0]-coord[3*E1+0], coord[3*E2+1]-coord[3*E1+1],
                     coord[3*E3+0]-coord[3*E2+0], coord[3*E3+1]-coord[3*E2+1]);
        float r = sqrt(cp1 * cp1 + cp2 * cp2 + cp3 * cp3);
        cp1 /= r;
        cp2 /= r;
        cp3 /= r;
        //printf("Vector for E%d is (%.3f, %.3f, %.3f)\n", E3, cp1, cp2, cp3);
        Force[3*E3] = -cp1 * rot[ii] * 1e-3;
        Force[3*E3+1] = -cp2 * rot[ii] * 1e-3;
        Force[3*E3+2] = -cp3 * rot[ii] * 1e-3;
    }
}
*/


