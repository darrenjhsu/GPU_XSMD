#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "kernel.cu"
#include "speedtest.hh"
#include "param.hh"
#include "coord_ref.hh"
//#include "raster8.hh"

int main () {
    hipFree(0); 
    float *d_Aq, *d_coord, *d_Force, *d_FF;
    int *d_Ele;
    float *d_q_S_ref_dS, *d_S_calc;
    float *S_calc1, *S_calc2;
    float *d_S_calcc, *d_f_ptxc, *d_f_ptyc, *d_f_ptzc;
    float *d_dx, *d_dy, *d_dz;
    float *d_raster, *d_V, *d_r2;
    float *d_WK;
    float *Force;
    int *d_close_flag, *d_close_num, *d_close_idx;
    float *d_vdW;
    int *close_num, *close_idx;
    float *V;
    float *d_FF_table;
    float *d_surf, *surf, *d_surf_grad;

    // set various memory chunk sizes
    int size_coord = 3 * num_atom * sizeof(float);
    int size_atom = num_atom * sizeof(int);
    int size_atom2 = num_atom2 * sizeof(int);
    int size_atom2f = num_atom2 * sizeof(float);
    int size_atomxatom = num_atom * num_atom * sizeof(float);
    int size_atom2xatom2 = num_atom2 * num_atom2 * sizeof(int);
    printf("size_atomxatom = %d. \n", size_atomxatom);
    int size_q = num_q * sizeof(float); 
    int size_FF = num_ele * num_q * sizeof(float);
    int size_qxatom2 = num_q2 * num_atom2 * sizeof(float); // check if overflow
    int size_raster = num_raster * 3 * sizeof(float);
    int size_FF_table = (num_ele+1) * num_q * sizeof(float);
    int size_surf = num_atom * num_raster * 3 * sizeof(float);
    int size_WK = 11 * num_ele * sizeof(float);
    int size_vdW = (num_ele+1) * sizeof(float);


    // Allocate local memories
    Force = (float *)malloc(size_coord);
    close_idx = (int *)malloc(size_atom2xatom2);
    close_num = (int *)malloc(size_atom2);
    V = (float *)malloc(size_atom2f);
    S_calc1 = (float *)malloc(size_q);
    S_calc2 = (float *)malloc(size_q);
    surf = (float *)malloc(size_surf);

    // Allocate cuda memories
    hipMalloc((void **)&d_Aq,     size_q);
    hipMalloc((void **)&d_coord,  size_coord); // 40 KB
    hipMalloc((void **)&d_Force,  size_coord); // 40 KB
    hipMalloc((void **)&d_FF,     size_FF);  // 10 KB ?
    hipMalloc((void **)&d_Ele,    size_atom);
    hipMalloc((void **)&d_q_S_ref_dS, 3 * size_q);
    hipMalloc((void **)&d_S_calc, size_q); // Will be computed on GPU
    hipMalloc((void **)&d_f_ptxc, size_qxatom2);
    hipMalloc((void **)&d_f_ptyc, size_qxatom2);
    hipMalloc((void **)&d_f_ptzc, size_qxatom2);
    hipMalloc((void **)&d_S_calcc, size_qxatom2);
    hipMalloc((void **)&d_V, size_atom2f);
    hipMalloc((void **)&d_dx, size_atomxatom);
    hipMalloc((void **)&d_dy, size_atomxatom);
    hipMalloc((void **)&d_dz, size_atomxatom);
    hipMalloc((void **)&d_r2, size_atomxatom);
    hipMalloc((void **)&d_close_flag, size_atom2xatom2);
    hipMalloc((void **)&d_close_num, size_atom2);
    hipMalloc((void **)&d_close_idx, size_atom2xatom2);
    hipMalloc((void **)&d_vdW, size_vdW);
    hipMalloc((void **)&d_FF_table, size_FF_table);
    hipMalloc((void **)&d_WK, size_WK);
    hipMalloc((void **)&d_surf, size_surf);
    hipMalloc((void **)&d_surf_grad, size_coord);
    // Initialize some matrices
    hipMemset(d_close_flag, 0, size_qxatom2);
    hipMemset(d_Force, 0.0, size_coord);
    hipMemset(d_Aq, 0.0, size_q);
    hipMemset(d_S_calc, 0.0, size_q);
    hipMemset(d_f_ptxc,0.0, size_qxatom2);
    hipMemset(d_f_ptyc,0.0, size_qxatom2);   
    hipMemset(d_f_ptzc,0.0, size_qxatom2);
    hipMemset(d_S_calcc,0.0, size_qxatom2);
    hipMemset(d_close_num, 0, size_atom2);
    hipMemset(d_close_idx, 0, size_atom2xatom2);
    hipMemset(d_surf, 0.0, size_surf);
    // Copy necessary data
    hipMemcpy(d_coord, coord_ref, size_coord,    hipMemcpyHostToDevice);
    hipMemcpy(d_vdW, vdW, size_vdW, hipMemcpyHostToDevice);
    hipMemcpy(d_Ele,    Ele,    size_atom,   hipMemcpyHostToDevice);
    hipMemcpy(d_q_S_ref_dS,  q_S_ref_dS, 3 * size_q,      hipMemcpyHostToDevice);
    hipMemcpy(d_WK,     WK,     size_WK,     hipMemcpyHostToDevice);

    float sigma2 = 1.0;
    float alpha = 1.0;
     
    dist_calc<<<1024, 1024>>>(d_coord, d_dx, d_dy, d_dz, d_r2, d_close_flag, num_atom, num_atom2); 
    pre_scan_close<<<2048,1024>>>(d_close_flag, d_close_num, d_close_idx, num_atom2);
    hipMemcpy(close_num, d_close_num, size_atom2, hipMemcpyDeviceToHost);
    surf_calc<<<1024,512>>>(d_coord, d_Ele, d_r2, d_close_num, d_close_idx, d_vdW, num_atom, num_atom2, num_raster, sol_s, d_V, d_surf, d_surf_grad, offset);
    sum_V<<<1,1024>>>(d_V, num_atom, num_atom2, d_Ele, d_vdW);
    FF_calc<<<320, 32>>>(d_q_S_ref_dS, d_WK, d_vdW, num_q, num_ele, c1, r_m, d_FF_table); 
    scat_calc<<<320, 1024>>>(d_coord,  d_Force,   d_Ele,     d_WK,     d_q_S_ref_dS, 
                             d_S_calc, num_atom,  num_q,     num_ele,  d_Aq, 
                             alpha,    k_chi,     sigma2,    d_f_ptxc, d_f_ptyc, 
                             d_f_ptzc, d_S_calcc, num_atom2, num_q2,   d_vdW,
                             c1,       c2,        d_V,       r_m,      d_FF_table,
                             d_surf_grad);
    hipMemcpy(S_calc1,d_S_calc, size_q,     hipMemcpyDeviceToHost);
    hipMemcpy(surf,   d_surf,   size_surf,  hipMemcpyDeviceToHost);
    force_calc<<<1024, 512>>>(d_Force, num_atom, num_q, d_f_ptxc, d_f_ptyc, d_f_ptzc, num_atom2, num_q2, d_Ele);
  
    // Initialize some matrices
    hipMemset(d_close_flag, 0, size_qxatom2);
    hipMemset(d_Force, 0.0, size_coord);
    hipMemset(d_Aq, 0.0, size_q);
    hipMemset(d_S_calc, 0.0, size_q);
    hipMemset(d_f_ptxc,0.0, size_qxatom2);
    hipMemset(d_f_ptyc,0.0, size_qxatom2);   
    hipMemset(d_f_ptzc,0.0, size_qxatom2);
    hipMemset(d_S_calcc,0.0, size_qxatom2);
    hipMemset(d_close_num, 0, size_atom2);
    hipMemset(d_close_idx, 0, size_atom2xatom2);
    hipMemset(d_surf, 0.0, size_surf);

    // Do the next structure
    hipMemcpy(d_coord, coord_init, size_coord,    hipMemcpyHostToDevice);

    dist_calc<<<1024, 1024>>>(d_coord, d_dx, d_dy, d_dz, d_r2, d_close_flag, num_atom, num_atom2); 
    pre_scan_close<<<2048,1024>>>(d_close_flag, d_close_num, d_close_idx, num_atom2);
    hipMemcpy(close_num, d_close_num, size_atom2, hipMemcpyDeviceToHost);
    surf_calc<<<1024,512>>>(d_coord, d_Ele, d_r2, d_close_num, d_close_idx, d_vdW, num_atom, num_atom2, num_raster, sol_s, d_V, d_surf, d_surf_grad, offset);
    sum_V<<<1,1024>>>(d_V, num_atom, num_atom2, d_Ele, d_vdW);
    FF_calc<<<320, 32>>>(d_q_S_ref_dS, d_WK, d_vdW, num_q, num_ele, c1, r_m, d_FF_table); 
    scat_calc<<<320, 1024>>>(d_coord,  d_Force,   d_Ele,     d_WK,     d_q_S_ref_dS, 
                             d_S_calc, num_atom,  num_q,     num_ele,  d_Aq, 
                             alpha,    k_chi,     sigma2,    d_f_ptxc, d_f_ptyc, 
                             d_f_ptzc, d_S_calcc, num_atom2, num_q2,   d_vdW,
                             c1,       c2,        d_V,       r_m,      d_FF_table,
                             d_surf_grad);
    hipMemcpy(S_calc2,d_S_calc, size_q,     hipMemcpyDeviceToHost);
    hipMemcpy(surf,   d_surf,   size_surf,  hipMemcpyDeviceToHost);
    force_calc<<<1024, 512>>>(d_Force, num_atom, num_q, d_f_ptxc, d_f_ptyc, d_f_ptzc, num_atom2, num_q2, d_Ele);
 

    printf("float q_S_ref_dS[%d] = {", 3*num_q);
    for (int ii = 0; ii < num_q; ii++) {
        printf("%f, ",q_S_ref_dS[ii]);
    }
    printf("\n");
    for (int ii = 0; ii < num_q; ii++) {
        printf("%f, ",S_calc2[ii]);
    }
    printf("\n");
    for (int ii = 0; ii < num_q; ii++) {
        printf("%f", S_calc1[ii]-S_calc2[ii]);
        if (ii < num_q - 1) printf(", ");
    }
    printf("};\n");
   
    for (int ii = 0; ii < num_q; ii ++) {
        printf("%f, ",S_calc2[ii]);
    } 

    // Calculating chi square
    /*
    float chi = 0.0;
    float chi2 = 0.0;
    float chi_ref = 0.0;
    for (int ii = 0; ii < num_q; ii++) {
        chi = q_S_ref_dS[ii+2*num_q] - (S_calc[ii] - q_S_ref_dS[ii+num_q]);
        printf("q = %.3f: chi is: %.3f, dS is: %.3f, S_calc is: %.3f, S_ref is: %.3f\n", q_S_ref_dS[ii], chi, q_S_ref_dS[ii+2*num_q], S_calc[ii], q_S_ref_dS[ii+num_q]); 
        chi2 += chi * chi;
        chi_ref+= q_S_ref_dS[ii+2*num_q] * q_S_ref_dS[ii+2*num_q];
    }
    printf("chi square is %.5e ( %.3f \% )\n", chi2, chi2 / chi_ref * 100);
    */

    // Print surface points for PDB exhibition
    /*
            printf("CRYST1    0.000    0.000    0.000  90.00  90.00  90.00 P 1           1\n");
    int idx = 0;
    for (int ii = 0; ii < num_atom * num_raster; ii++) {
        if (surf[3*ii] != 0) {
            printf("ATOM  %5d  XXX XXX P   1     %7.3f %7.3f %7.3f  0.00  0.00      P1\n", idx, surf[3*ii], surf[3*ii+1], surf[3*ii+2]);
            idx++;
        }
    }
    */

    // Free cuda and local memories
    hipFree(d_coord); hipFree(d_Force); 
    hipFree(d_Ele); hipFree(d_FF); 
    hipFree(d_q_S_ref_dS); 
    hipFree(d_S_calc); hipFree(d_Aq);
    hipFree(d_f_ptxc); hipFree(d_f_ptyc); hipFree(d_f_ptzc);
    hipFree(d_S_calcc); hipFree(d_WK);
    hipFree(d_dx); hipFree(d_dy); hipFree(d_dz);
    hipFree(d_V); 
    hipFree(d_r2);
    hipFree(d_close_flag); hipFree(d_close_num); hipFree(d_close_idx);
    hipFree(d_vdW);
    free(S_calc1); free(S_calc2); free(close_num); free(close_idx);

    return 0;
}
